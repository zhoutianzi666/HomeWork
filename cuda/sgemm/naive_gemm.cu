
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"

#define WARMUP 10
#define REPEATE 10

using DATATYPE = half;
#define DATATYPE_BYTE 2

using ACCU_DATATYPE = float;
#define ACCU_DATATYPE_BYTE 4

__global__ void matmul_gpu1(DATATYPE *a, DATATYPE *b, DATATYPE *c, int m, int n,
                            int k) {
  const int tidx = threadIdx.x;
  const int bidx = blockIdx.x;
  int idx = tidx + bidx * blockDim.x;
  const int row = idx / n;
  const int col = idx % n;

  if (row >= m || col >= n) return;

  ACCU_DATATYPE sum = 0.;
  for (int i = 0; i < k; i++) {
#if DATATYPE_BYTE == 4
    sum += a[row * k + i] * b[i * n + col];
#elif DATATYPE_BYTE == 2
    sum += __half2float(a[row * k + i] * b[i * n + col]);
#endif
  }

#if DATATYPE_BYTE == 4
  c[row * n + col] = sum;
#elif DATATYPE_BYTE == 2
  c[row * n + col] = __float2half(sum);
#endif
}

#define block_K 512
__global__ void matmul_gpu2(DATATYPE *a, DATATYPE *b, DATATYPE *c, int m,
                            int n, int k) {
  const int tidx = threadIdx.x;
  const int bidx = blockIdx.x;
  int idx = tidx + bidx * blockDim.x;
  const int row = idx / n;
  const int col = idx % n;
  __shared__ DATATYPE aTile[block_K];

  if (row >= m || col >= n) return;

  ACCU_DATATYPE sum = 0.;

  for (int i = 0; i < k; i += block_K) {
    if (tidx < block_K && tidx + i < k) {
      aTile[tidx] = a[row * k + tidx + i];
    }

    __syncthreads();

    for (int j = i; j < i + block_K; j++) {
#if DATATYPE_BYTE == 4
      sum += aTile[j - i] * b[j * n + col];
#elif DATATYPE_BYTE == 2
      sum += __half2float(aTile[j - i] * b[j * n + col]);
#endif
    }
    __syncthreads();
  }

#if DATATYPE_BYTE == 4
  c[row * n + col] = sum;
#elif DATATYPE_BYTE == 2
  c[row * n + col] = __float2half(sum);
#endif

}

int main(void) {
  int m = 512;
  int n = 512;
  int k = 512;
  DATATYPE *a, *b;
  hipError_t status = hipHostMalloc(&a, sizeof(DATATYPE) * m * k, hipHostMallocDefault);
  if (status != hipSuccess) {
    printf("分配内存失败");
  }
  status = hipHostMalloc(&b, sizeof(DATATYPE) * k * n, hipHostMallocDefault);
  if (status != hipSuccess) {
    printf("分配内存失败");
  }
  for (int i = 0; i < m * k; i++) {
#if DATATYPE_BYTE == 4
    a[i] = (rand() % 9999) / 10000.0;
#else
    a[i] = __float2half((rand() % 9999) / 10000.0 - 0.5);
#endif
  }
  for (int i = 0; i < k * n; i++) {
#if DATATYPE_BYTE == 4
    b[i] = (rand() % 9999) / 10000.0;
#else
    b[i] = __float2half((rand() % 9999) / 10000.0 - 0.5);
#endif
  }

  DATATYPE *c;
  hipHostMalloc(&c, sizeof(DATATYPE) * m * n, hipHostMallocDefault);
  memset(c, 0, sizeof(DATATYPE) * m * n);

  float *c_cpu_fp32 = (float *)malloc(sizeof(float) * m * n);
  memset(c_cpu_fp32, 0, sizeof(float) * m * n);

  DATATYPE *dev_a, *dev_b;
  DATATYPE *dev_c;

  // allocate the memory on the GPU
  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();

  hipMalloc((void **)&dev_a, m * k * sizeof(DATATYPE));
  hipMalloc((void **)&dev_b, k * n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_c, m * n * sizeof(DATATYPE));

  hipMemcpy(dev_a, a, m * k * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, k * n * sizeof(DATATYPE), hipMemcpyHostToDevice);

  uint3 grid = {m * n / 512 + 1, 1, 1};
  uint3 block = {512, 1, 1};

  for (int i = 0; i < WARMUP; i++) {
    matmul_gpu2<<<grid, block, 0 * block_K * sizeof(DATATYPE)>>>(
        dev_a, dev_b, dev_c, m, n, k);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {
    matmul_gpu2<<<grid, block, 0 * block_K * sizeof(DATATYPE)>>>(
        dev_a, dev_b, dev_c, m, n, k);
  }

  hipEventRecord(end);
  hipEventSynchronize(beg);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("%f\n", elapsed_time);

  hipMemcpy(c, dev_c, m * n * sizeof(DATATYPE), hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      double sum = 0.f;
      for (int ii = 0; ii < k; ii++) {
#if DATATYPE_BYTE == 4
        sum += a[i * k + ii] * b[ii * n + j];
#else
        sum += __half2float(a[i * k + ii]) * __half2float(b[ii * n + j]);
#endif
      }
      c_cpu_fp32[i * n + j] = sum;
    }
  }

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  double max_diff = -1.;
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
#if DATATYPE_BYTE == 4
      double c_gpu_fp32 = c[i * n + j];
#else
      double c_gpu_fp32 = __half2float(c[i * n + j]);
#endif
      if (std::abs(c_cpu_fp32[i * n + j] - c_gpu_fp32) > max_diff) {
        max_diff = std::abs(c_cpu_fp32[i * n + j] - c_gpu_fp32);
      }
    }
  }

  printf("%f\n", max_diff);

  hipDeviceReset();
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  free(c_cpu_fp32);
  return 0;
}