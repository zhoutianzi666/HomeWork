#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"

#define CUDA_CHECK(stmt)                                  \
do {                                                      \
    hipError_t result = (stmt);                          \
    if (hipSuccess != result) {                          \
        fprintf(stderr, "[%s:%d] CUDA failed with %s \n", \
         __FILE__, __LINE__, hipGetErrorString(result)); \
        exit(-1);                                         \
    }                                                     \
} while (0)

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main (int argc, char *argv[]) {
    int mype_node, msg;
    hipStream_t stream;
    int rank, nranks;
    nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
    nvshmemx_uniqueid_t id = NVSHMEMX_UNIQUEID_INITIALIZER;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    
    printf("rank %d， nranks %d\n", rank, nranks);

    // PE 0 queries the unique ID
    if (rank == 0) {
       nvshmemx_get_uniqueid(&id);
    }

    // PE 0 broadcast the unique ID to all peers
    MPI_Bcast(&id, sizeof(nvshmemx_uniqueid_t), MPI_UINT8_T, 0, MPI_COMM_WORLD);
    nvshmemx_set_attr_uniqueid_args(rank, nranks, &id, &attr);
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);
    // mype_node 应该是我这台机器的上面的node的局部id！
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    int *destination = (int *) nvshmem_malloc (sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    CUDA_CHECK(hipMemcpyAsync(&msg, destination, sizeof(int),
                hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}